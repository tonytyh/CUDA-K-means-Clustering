
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
This example demonstrates how to use the Cuda OpenGL bindings to
dynamically modify a vertex buffer using a Cuda kernel.

The steps are:
1. Create an empty vertex buffer object (VBO)
2. Register the VBO with Cuda
3. Map the VBO for writing from Cuda
4. Run Cuda kernel to modify the vertex positions
5. Unmap the VBO
6. Render the results using OpenGL

Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#if defined (__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#ifndef glutCloseFunc
#define glutCloseFunc glutWMCloseFunc
#endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;


float g_fAnim = 0.0;

//define vertex

struct SVertex
{
	GLfloat x, y, z;
	GLfloat r, g, b;
};

SVertex * Vertices;


// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
	unsigned int vbo_res_flags);


// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);
const char *sSDKsample = "simpleGL (VBO)";


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
	// Create the CUTIL timer
	sdkCreateTimer(&timer);

	// First initialize OpenGL context, so we can properly set the GL for CUDA.
	// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	if (false == initGL(&argc, argv))
	{
		return false;
	}

	cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());

	// register callbacks
	glutMouseFunc(mouse);

#if defined (__APPLE__) || defined(MACOSX)
	atexit(cleanup);
#else
	glutCloseFunc(cleanup);
#endif

	// create Vertices
	Vertices = new SVertex[10000];

	for (int i = 0; i < 10000; i++)
	{
		Vertices[i].x = float(rand()) / RAND_MAX;
		Vertices[i].y = float(rand()) / RAND_MAX;
		Vertices[i].z = float(rand()) / RAND_MAX;

		Vertices[i].r = float(rand()) / RAND_MAX;
		Vertices[i].g = float(rand()) / RAND_MAX;
		Vertices[i].b = float(rand()) / RAND_MAX;
	}
	// start rendering mainloop
	glutMainLoop();
	
	return 0;
}

void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}

	char fps[256];
	sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
	glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Cuda GL Interop (VBO)");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMotionFunc(motion);
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

	// initialize necessary OpenGL extensions
	if (!isGLVersionSupported(2, 0))
	{
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.1, 10.0);

	SDK_CHECK_ERROR_GL();

	return true;
}



#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif



////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
	sdkStartTimer(&timer);

	// run CUDA kernel to generate vertex positions
	//runCuda(&cuda_vbo_resource);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, translate_z);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);

	// render from the vbo
	//glBindBuffer(GL_ARRAY_BUFFER, vbo);
	//glVertexPointer(4, GL_FLOAT, 0, 0);
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);

	glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);

	glVertexPointer(3,   //3 components per vertex (x,y,z)
		GL_FLOAT,
		sizeof(SVertex),
		Vertices);
	//pass the color pointer
	glColorPointer(3,   //3 components per vertex (r,g,b)
		GL_FLOAT,
		sizeof(SVertex),
		&Vertices[0].r);  //Pointer to the first color
						  //point size for point mode (press p for that one)
	glPointSize(2.0);
	//glClearColor(0.0, 0.0, 0.0, 0.0);


	//glColor3f(1.0, 0.0, 0.0);
	//glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
	glDrawArrays(GL_POINTS, 0, 10000);
	glDisableClientState(GL_VERTEX_ARRAY);

	glutSwapBuffers();

	g_fAnim += 0.01f;

	sdkStopTimer(&timer);
	computeFPS();
}

void timerEvent(int value)
{
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	}
}

void cleanup()
{
	sdkDeleteTimer(&timer);


}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case (27):
#if defined(__APPLE__) || defined(MACOSX)
		exit(EXIT_SUCCESS);
#else
		glutDestroyWindow(glutGetWindow());
		return;
#endif
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1 << button;
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

void motion(int x, int y)
{
	float dx, dy;
	dx = (float)(x - mouse_old_x);
	dy = (float)(y - mouse_old_y);

	if (mouse_buttons & 1)
	{
		rotate_x += dy * 0.2f;
		rotate_y += dx * 0.2f;
	}
	else if (mouse_buttons & 4)
	{
		translate_z += dy * 0.01f;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}








