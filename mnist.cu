#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
This example demonstrates how to use the Cuda OpenGL bindings to
dynamically modify a vertex buffer using a Cuda kernel.

The steps are:
1. Create an empty vertex buffer object (VBO)
2. Register the VBO with Cuda
3. Map the VBO for writing from Cuda
4. Run Cuda kernel to modify the vertex positions
5. Unmap the VBO
6. Render the results using OpenGL

Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include<random>
#include<iostream>
#include<fstream>
#include<sstream>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#if defined (__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#ifndef glutCloseFunc
#define glutCloseFunc glutWMCloseFunc
#endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>


//sleep
//#include <pthread>
//#include <unistd>




#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 1024 * 2;
const unsigned int window_height = 1024;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

long number_of_data = 600000;
const long number_of_clusters = 10;




float g_fAnim = 0.0;

//define vertex

struct SVertex
{
	GLfloat x, y, z;
	GLfloat r, g, b;
};

typedef struct
{
	float x = 0;
	float y = 0;
	float z = 0;
	long c = NULL;
}point;

typedef struct
{
	float r, g, b;
}Color;

SVertex * Vertices;
SVertex * Vertices_centroids;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
	unsigned int vbo_res_flags);


// rendering callbacks
void display();
void display_centroids();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);
const char *sSDKsample = "simpleGL (VBO)";



//kmeans part

void update_centroids(point * data, point * centroids, long *cluster_volume, long number_of_data, const long number_of_clusters)
{

	//point temp_sum[number_of_clusters];
	point * temp_sum = new point[number_of_clusters];
	//memset(cluster_volume, 0, sizeof(long) * number_of_clusters);
	for (int c = 0; c < number_of_clusters; c++)
	{
		temp_sum[c].x = 0;
		temp_sum[c].y = 0;
		temp_sum[c].z = 0;
		cluster_volume[c] = 0;
	}

	for (long i = 0; i < number_of_data; i++)
	{
		temp_sum[data[i].c].x += data[i].x;
		temp_sum[data[i].c].y += data[i].y;
		temp_sum[data[i].c].z += data[i].z;
		cluster_volume[data[i].c]++;

	}

	for (long c = 0; c < number_of_clusters; c++)
		//average
		if (cluster_volume[c] != 0)
		{
			temp_sum[c].x /= cluster_volume[c];
			temp_sum[c].y /= cluster_volume[c];
			temp_sum[c].z /= cluster_volume[c];
		}

	for (long c = 0; c < number_of_clusters; c++)
	{
		centroids[c].x = temp_sum[c].x;
		centroids[c].y = temp_sum[c].y;
		centroids[c].z = temp_sum[c].z;
	}


}



void print_data(point * data, long number_of_data)
{
	for (long i = 0; i<number_of_data; i++)
	{
		printf("[%.3f,%.3f,%.3f %d]\n", data[i].x, data[i].y, data[i].z, data[i].c);
	}

}
void print_centroids(point * centroids, long number_of_clusters, long * cluster_volume)
{
	for (long c = 0; c < number_of_clusters; c++)
	{
		printf("centroids: %d [%.3f,%.3f,%.3f] volume: %d\n", c, centroids[c].x, centroids[c].y, centroids[c].z, cluster_volume[c]);
	}

}


__global__ void  update_cluster_label(point * d_data, point * d_centroids, long d_number_of_data, long d_number_of_clusters)
{
	//global index
	long idx = blockDim.x * blockIdx.x + threadIdx.x;
	float min = 10000000.0;
	float temp;
	//	long new_label = NULL;

	if (idx < d_number_of_data)
	{
		for (long c = 0; c < d_number_of_clusters; c++)
		{
			temp = sqrt(pow(d_data[idx].x - d_centroids[c].x, 2) + pow(d_data[idx].y - d_centroids[c].y, 2) + pow(d_data[idx].z - d_centroids[c].z, 2));
			if (temp < min)
			{
				min = temp;
				d_data[idx].c = c;
			}

		}
	}

}

//convert data to vertex

void data2vertex(point * data, SVertex * Vertices, Color * color_table)
{
	for (int i = 0; i < number_of_data; i++)
	{
		Vertices[i].x = data[i].x;
		Vertices[i].y = data[i].y;
		Vertices[i].z = data[i].z;

		Vertices[i].r = color_table[data[i].c].r;
		Vertices[i].g = color_table[data[i].c].g;
		Vertices[i].b = color_table[data[i].c].b;

	}
}


void centroids2vertex(point * centroids, SVertex * Vertices)
{
	for (int i = 0; i < number_of_clusters; i++)
	{
		Vertices[i].x = centroids[i].x;
		Vertices[i].y = centroids[i].y;
		Vertices[i].z = centroids[i].z;

		Vertices[i].r = 1;
		Vertices[i].g = 1;
		Vertices[i].b = 1;

	}
}

void generate_random_centroids(point * centroids, long number_of_clusters)
{
	srand(time(0));
	for (long i = 0; i < number_of_clusters; i++)
	{
		centroids[i].x = float(rand()) / RAND_MAX;
		centroids[i].y = float(rand()) / RAND_MAX;
		centroids[i].z = float(rand()) / RAND_MAX;
	}
}


void read_mnist_centroids(point * centroids)
{
	char mnist_centroids_path[] = "C:\\Users\\yut107\\Desktop\\GPU_project\\class_labs(1)\\class_labs\\Src\\mnist\\centroids_kf.txt";

	std::ifstream file(mnist_centroids_path);
	std::string linebuffer;

	int number_of_clusters = 10;
	for (int i = 0; i < number_of_clusters; i++)
	{
		getline(file, linebuffer);
		std::istringstream line(linebuffer);
		float x, y, z;
		line >> x >> y >> z;
		centroids[i].x = x + float(rand()) / RAND_MAX * 0.1;
		centroids[i].y = y + float(rand()) / RAND_MAX * 0.1;
		centroids[i].z = z + float(rand()) / RAND_MAX * 0.1;
		//std::cout << x << y << z << std::endl;
	}

}

long  read_mnist_data(point * data)
{
	char mnist_data_path[] = "C:\\Users\\yut107\\Desktop\\GPU_project\\class_labs(1)\\class_labs\\Src\\mnist\\data_label_kf.txt";

	std::ifstream file(mnist_data_path);
	std::string linebuffer;

	int number_of_clusters = 10;
	long  counter = 0;
	while (file && getline(file, linebuffer))
	{
		std::istringstream line(linebuffer);
		float x, y, z, c;
		line >> x >> y >> z >> c;
		data[counter].x = x;
		data[counter].y = y;
		data[counter].z = z;
		data[counter].c = (int)c;
		counter++;
		//std::cout << x << y << z << std::endl;
	}

	return counter;

}





////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
	// Create the CUTIL timer
	sdkCreateTimer(&timer);
	srand(time(0));


	long cluster_volume[number_of_clusters] = { 0 };
	Color color_table[number_of_clusters];

	//create color table

	for (int c = 0; c < number_of_clusters; c++)
	{
		color_table[c].r = float(rand()) / RAND_MAX;
		color_table[c].g = float(rand()) / RAND_MAX;
		color_table[c].b = float(rand()) / RAND_MAX;
		printf("[%d] r: %.3f g: %.3f b: %.3f\n", c, color_table[c].r, color_table[c].g, color_table[c].b);
	}

	// read centroids from mnist
	point centroids[number_of_clusters];
	read_mnist_centroids(centroids);

	//read data from mnist
	point * temp_data = new point[1000000];
	number_of_data = read_mnist_data(temp_data);
	point * data = new point[number_of_data];
	memcpy(data, temp_data, sizeof(point) * number_of_data);
	delete[] temp_data;
	
	
	//generate_random_centroids(centroids, number_of_clusters);


	//generate random color for each point
	for (int i = 0; i < number_of_data; i++)
	{
		int c = rand() % number_of_clusters;
		data[i].c = c;
		cluster_volume[c]++;
	}

	////for (int c = 0; c < number_of_clusters; c++)
	////	printf("[%d]: %d\n", c, cluster_volume[c]);

	//// generate normal distribution
	//std::default_random_engine de(time(0));
	////std::normal_distribution<float> nd_x(0, 0.08);
	////std::normal_distribution<float> nd_y(0, 0.08);
	////std::normal_distribution<float> nd_z(0, 0.08);
	//std::normal_distribution<float> nd_x(0, float(rand()) / RAND_MAX);
	//std::normal_distribution<float> nd_y(0, float(rand()) / RAND_MAX);
	//std::normal_distribution<float> nd_z(0, float(rand()) / RAND_MAX);
	//for (int i = 0; i < number_of_data; i++)
	//{
	//	int c = rand() % number_of_clusters;
	//	data[i].x = centroids[c].x + nd_x(de);
	//	data[i].y = centroids[c].y + nd_y(de);
	//	data[i].z = centroids[c].z + nd_z(de);
	//}


	//	define cuda data
	point * d_data;
	hipMalloc(&d_data, sizeof(point) * number_of_data);
	hipMemcpy(d_data, data, sizeof(point) * number_of_data, hipMemcpyHostToDevice);


	//define cuda cnetroids
	point * d_centroids;
	hipMalloc(&d_centroids, sizeof(point) * number_of_clusters);
	hipMemcpy(d_centroids, centroids, sizeof(point) * number_of_clusters, hipMemcpyHostToDevice);

	//	print_data(data, number_of_data);
	print_centroids(centroids, number_of_clusters, cluster_volume);








	// First initialize OpenGL context, so we can properly set the GL for CUDA.
	// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	if (false == initGL(&argc, argv))
	{
		return false;
	}

	cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());

	// register callbacks
	glutMouseFunc(mouse);

#if defined (__APPLE__) || defined(MACOSX)
	atexit(cleanup);
#else
	glutCloseFunc(cleanup);
#endif

	// create Vertices
	Vertices = new SVertex[number_of_data];
	Vertices_centroids = new SVertex[number_of_clusters];

	//replay buffer 

	//SVertex Replay[20][number_of_data];

	// start rendering mainloop
	data2vertex(data, Vertices, color_table);
	display();
	Sleep(1000);

	for (long i = 0; i < 10; i++)
	{
		//update

		data2vertex(data, Vertices, color_table);
		centroids2vertex(centroids, Vertices_centroids);
		display();
		Sleep(100);
		//display_centroids();
		update_cluster_label << <number_of_data / 1024 + 1, 1024 >> > (d_data, d_centroids, number_of_data, number_of_clusters);
		hipMemcpy(data, d_data, sizeof(point) * number_of_data, hipMemcpyDeviceToHost);
		update_centroids(data, centroids, cluster_volume, number_of_data, number_of_clusters);
		printf("iteration %d:\n\n", i);
		print_centroids(centroids, number_of_clusters, cluster_volume);
		hipMemcpy(d_centroids, centroids, sizeof(point) * number_of_clusters, hipMemcpyHostToDevice);
		//printf("%d", time(0));

	}


	// start rendering mainloop
	glutMainLoop();

	return 0;
}

void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}

	char fps[256];
	sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
	glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Cuda GL Interop (VBO)");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMotionFunc(motion);
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

	// initialize necessary OpenGL extensions
	if (!isGLVersionSupported(2, 0))
	{
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.1, 10.0);

	SDK_CHECK_ERROR_GL();

	return true;
}



#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif



////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
	sdkStartTimer(&timer);

	// run CUDA kernel to generate vertex positions
	//runCuda(&cuda_vbo_resource);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, translate_z);
	//glTranslatef(0.0, 0.0, 0);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);


	// render from the vbo
	//glBindBuffer(GL_ARRAY_BUFFER, vbo);
	//glVertexPointer(4, GL_FLOAT, 0, 0);
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);

	glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);

	glVertexPointer(3,   //3 components per vertex (x,y,z)
		GL_FLOAT,
		sizeof(SVertex),
		Vertices);
	//pass the color pointer
	glColorPointer(3,   //3 components per vertex (r,g,b)
		GL_FLOAT,
		sizeof(SVertex),
		&Vertices[0].r);  //Pointer to the first color
						  //point size for point mode (press p for that one)


	glPointSize(2);
	glDrawArrays(GL_POINTS, 0, number_of_data);

	glVertexPointer(3,   //3 components per vertex (x,y,z)
		GL_FLOAT,
		sizeof(SVertex),
		Vertices_centroids);
	//pass the color pointer
	glColorPointer(3,   //3 components per vertex (r,g,b)
		GL_FLOAT,
		sizeof(SVertex),
		&Vertices_centroids[0].r);

	glPointSize(4.0);
	glDrawArrays(GL_POINTS, 0, number_of_clusters);



	glDisableClientState(GL_VERTEX_ARRAY);
	glutSwapBuffers();



	g_fAnim += 0.01f;

	sdkStopTimer(&timer);
	computeFPS();
	//glutGet(5000);

}




void timerEvent(int value)
{
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	}
}

void cleanup()
{
	sdkDeleteTimer(&timer);


}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case (27):
#if defined(__APPLE__) || defined(MACOSX)
		exit(EXIT_SUCCESS);
#else
		glutDestroyWindow(glutGetWindow());
		return;
#endif
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1 << button;
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

void motion(int x, int y)
{
	float dx, dy;
	dx = (float)(x - mouse_old_x);
	dy = (float)(y - mouse_old_y);

	if (mouse_buttons & 1)
	{
		rotate_x += dy * 0.2f;
		rotate_y += dx * 0.2f;
	}
	else if (mouse_buttons & 4)
	{
		translate_z += dy * 0.01f;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}








